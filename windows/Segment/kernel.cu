#include "Stdafx.h"
#include "HFSSegment.h"

int main()
{
	hipSetDevice(0);

	// run on BSDS dataset
	/*DataSet bsds("C:/WkDir/BSR/BSDS500/");
	HFSSegment engine(bsds);
    engine.runDataSet(0.28f, 200);*/

	// segment a single image
	HFSSegment engine;
	Mat img3u = imread("C:/WkDir/BSR/BSDS500/JPEGImages/103078.jpg");
	Mat seg, show;
	int num_css = engine.processImage(seg, img3u, 0.28f, 200);
	engine.drawSegmentationRes(show, seg, img3u, num_css);
	imshow("Segmentation", show);
	waitKey(0);

	return 0;
}

